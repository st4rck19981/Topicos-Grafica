#include "hip/hip_runtime.h"
//https://fisica.cab.cnea.gov.ar/gpgpu/images/charlas/partec%20-%20programacinbasicoii.pdf

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void SumaColMatrizKernel(int M, float* Md, float* Nd) {
	__shared__ float Nds[512];
	float Pvalue = 0;
	int aux = blockIdx.y * M + threadIdx.x * (M / blockDim.x);
	int aux2 = aux + (M / blockDim.x);
	for (int k = aux; k < aux2; ++k) {
		Pvalue = Pvalue + Md[k];
	}
	Nds[threadIdx.x] = Pvalue;
	__syncthreads();

	if (threadIdx.x == 0) {
		for (int i = 1; i < blockDim.x; ++i) {
			Nds[0] = Nds[0] + Nds[i];
		}
		Nd[blockIdx.y] = Nds[0];
	}
}


int main() {
	int f = 1024, c = 512;

	int size = f * c * sizeof(float);
	int size2 = c * sizeof(float);

	float* Mh = (float*)malloc(size);
	float* Nh = (float*)malloc(size2);

	for (int i = 0; i < f * c; i++)    Mh[i] = 1.0;

	float* Md, * Nd;
	hipMalloc(&Md, size);
	hipMalloc(&Nd, size2);

	hipMemcpy(Md, Mh, size, hipMemcpyHostToDevice);
	hipMemset(Nd, 0, size2);

	int M = f;
	int N = c;

	int chunk = 32;
	dim3 tamGrid(1, N);
	dim3 tamBlock(M / chunk, 1, 1);

	SumaColMatrizKernel << <tamGrid, tamBlock >> > (M, Md, Nd);
	hipMemcpy(Nh, Nd, size2, hipMemcpyDeviceToHost);

	hipFree(Md);
	hipFree(Nd);

	cout << "\nResultados: " << endl;
	for (int i = 0; i < c; i++) {
		cout << Nh[i] << " ";
	}

	return 0;
}
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "Funciones.h"
#include <iostream>
using namespace std;

#define Mask_size 3
#define TILE_SIZE 32
#define BLOCK_SIZE 32

__device__ unsigned char clamp(int value)//__device__ because it's called by a kernel
{
	if (value < 0)
		value = 0;
	else
		if (value > 255)
			value = 255;
	return  value;
}

__global__ void CUDA(unsigned char* In, char* Mask, unsigned char* Out, int Mask_Width, int Rowimg, int Colimg) {
	unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

	int Pvalue = 0;
	double suma = 0;

	int N_start_point_row = row - (Mask_Width / 2);
	int N_start_point_col = col - (Mask_Width / 2);

	for (int i = 0; i < Mask_Width; i++){
		for (int j = 0; j < Mask_Width; j++){
			if ((N_start_point_col + j >= 0 && N_start_point_col + j < Rowimg)
				&& (N_start_point_row + i >= 0 && N_start_point_row + i < Colimg))			{
				Pvalue += In[(N_start_point_row + i) * Rowimg + (N_start_point_col + j)] * Mask[i * Mask_Width + j];
			}
			else {
				Pvalue = 0;
			}
		}
	}
	suma = Pvalue;
	Out[row * Rowimg + col] = clamp(suma);
}

void callKernel2D(unsigned char* img, unsigned char* imgOut,char* h_Mask,int Mask_Width, int Row, int Col) {
//problemas con mask_size?
	int Size_of_bytes = sizeof(unsigned char) * Row * Col;
	int Mask_size_bytes = sizeof(char) * (Mask_size * Mask_size);
	unsigned char* d_In, * d_Out;
	char* d_Mask;
	float Blocksize = BLOCK_SIZE;

	hipMalloc((void**)&d_In, Size_of_bytes);
	hipMalloc((void**)&d_Out, Size_of_bytes);
	hipMalloc((void**)&d_Mask, Mask_size_bytes);

	hipMemcpy(d_In, img, Size_of_bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_Mask, h_Mask, Mask_size_bytes, hipMemcpyHostToDevice);

	dim3 dimGrid(ceil(Row / Blocksize), ceil(Col / Blocksize), 1);
	dim3 dimBlock(Blocksize, Blocksize, 1);

	CUDA<< <dimGrid, dimBlock >> > (d_In, d_Mask, d_Out, Mask_Width, Row, Col);

	hipDeviceSynchronize();
	hipMemcpy(imgOut, d_Out, Size_of_bytes, hipMemcpyDeviceToHost);

	hipFree(d_In);
	hipFree(d_Out);
	hipFree(d_Mask);
}
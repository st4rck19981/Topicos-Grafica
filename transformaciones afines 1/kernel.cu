#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "Funciones.h"
using namespace std;


#define Mask_size 3
#define TILE_SIZE 32
#define BLOCK_SIZE 32

__global__ void gpu_matrix_mult(int* a, int* b, int* c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (col < k && row < m)
    {
        for (int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}

void llamadakernel(int matriz[2][3], int matrix2[3][1], int matrixresult[2][1])
{
    int* dev_a;
    int* dev_b;
    int* dev_c;
    int size_a = 2 * 3 * sizeof(int);
    int size_b = 3 * 1 * sizeof(int);
    int size_c = 2 * 1 * sizeof(int);

    hipMalloc((void**)&dev_a, size_a);
    hipMalloc((void**)&dev_b, size_b);
    hipMalloc((void**)&dev_c, size_c);

    hipMemcpy(dev_a, matriz, size_a, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, matrix2, size_b, hipMemcpyHostToDevice);

    int numbloques = 1;
    int tambloque = 2;

    gpu_matrix_mult << <numbloques, tambloque >> > (dev_a, dev_b, dev_c, 2, 3, 1);

    hipDeviceSynchronize();
    hipMemcpy(matrixresult, dev_c, sizeof(int) * size_c, hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);


}
